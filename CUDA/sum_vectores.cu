#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void Vec_add(float *c, float *a, float *b, int n) {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < n){
     c[idx] = a[idx] + b[idx];
   }
}

int main(int argc, char* argv[]) {
   float *a_h, *b_h, *c_h;
   float *a_d, *b_d, *c_d;
   const int n=24;
   size_t size=n *sizeof(float);

   a_h = (float*) malloc(size);
   b_h = (float*) malloc(size);
   c_h = (float*) malloc(size);

   for (int i = 0; i < n; i++) {
      a_h[i] = i;
      b_h[i] = 	i+1;
   }

   printf("\nArreglo a:\n");
   for(int i=0;i<n;i++)
   		printf("%f ",a_h[i]);
   printf("\nArreglo b:\n");
   for(int i=0;i<n;i++)
   		printf("%f ",b_h[i]);
   
   hipMalloc((void**)&a_d, size);
   hipMalloc((void**)&b_d, size);
   hipMalloc((void**)&c_d, size);

   hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
   hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

   int block_size=8;
   int n_blocks=n/block_size+(n%block_size==0 ? 0:1);
   Vec_add<<<n_blocks,block_size>>>(c_d, a_d, b_d, n);

   hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

   printf("\nLa suma es: \n");
   for (int i = 0; i < n; i++)
      printf("%f ", c_h[i]);
   printf("\n");

   
   free(a_h);
   free(b_h);
   free(c_h);
   hipFree(a_d);
   hipFree(b_d);
   hipFree(c_d);
   return 0;
}